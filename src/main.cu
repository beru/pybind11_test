#include "hip/hip_runtime.h"

#include <pybind11/pybind11.h>
#include <dlpack.h>

#define STRINGIFY(x) #x
#define MACRO_STRINGIFY(x) STRINGIFY(x)

namespace py = pybind11;

void print_info(const DLTensor* tensor)
{
    printf("context : %d %d\n", tensor->ctx.device_type, tensor->ctx.device_id);
    printf("dtype : %d %d %d\n", tensor->dtype.code, tensor->dtype.bits, tensor->dtype.lanes);
    printf("ndim : %d\n", tensor->ndim);
    printf("shape : ");
    int64_t len = 1;
    for (int i=0; i<tensor->ndim; ++i) {
        len *= tensor->shape[i];
        printf("%ld ", tensor->shape[i]);
    }
    printf("\n");
    printf("strides : ");
    for (int i=0; i<tensor->ndim; ++i) {
        printf("%ld ", tensor->strides[i]);
    }
    printf("\n");
    printf("byte_offset : %lu\n", tensor->byte_offset);
}

int add(int i, int j) {
    return i + j;
}

__global__
void VecAdd(float* data, size_t length, float value)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= length)
        return;
    data[idx] += value + idx;
}

void add_gpu(py::handle capsule, float value)
{
    DLTensor* tensor = (DLTensor*) PyCapsule_GetPointer(capsule.ptr(), "dltensor");
    // print_info(tensor);
    size_t len = 1;
    for (int i=0; i<tensor->ndim; ++i) {
        len *= tensor->shape[i];
    }
    // printf("len : %ld\n", len);
    // printf("value : %f\n", value);
    VecAdd<<<(len+127)/128, 128>>>((float*)tensor->data, len, value);
}

PYBIND11_MODULE(pybind11_test, m) {
    m.doc() = R"pbdoc(
        Pybind11 example plugin
        -----------------------
        .. currentmodule:: pybind11_test
        .. autosummary::
           :toctree: _generate
           add
           subtract
    )pbdoc";

    m.def("add", &add, R"pbdoc(
        Add two numbers
        Some other explanation about the add function.
    )pbdoc");

    m.def("subtract", [](int i, int j) { return i - j; }, R"pbdoc(
        Subtract two numbers
        Some other explanation about the subtract function.
    )pbdoc");

    m.def("add_gpu", &add_gpu, R"pbdoc(
        Add value to DLPack tensor data
        Some other explanation about the add_gpu function is blowing in the wind.
    )pbdoc");


#ifdef VERSION_INFO
    m.attr("__version__") = MACRO_STRINGIFY(VERSION_INFO);
#else
    m.attr("__version__") = "dev";
#endif
}

